#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void xor_kernel(const uint8_t* in, uint8_t* out, size_t n)
{
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) out[i] = in[i] ^ 0x5A;
}

extern "C"
size_t dummy_gpu_xor(const void* src, size_t len, char** dst_out)
{
  const uint8_t* h_src = static_cast<const uint8_t*>(src);
  uint8_t* d_src;  hipMalloc(&d_src, len);
  uint8_t* d_dst;  hipMalloc(&d_dst, len);

  hipMemcpy(d_src, h_src, len, hipMemcpyHostToDevice);

  int block = 256;
  int grid  = (len + block - 1) / block;
  xor_kernel<<<grid, block>>>(d_src, d_dst, len);

  uint8_t* h_dst = (uint8_t*)malloc(len);
  hipMemcpy(h_dst, d_dst, len, hipMemcpyDeviceToHost);

  hipFree(d_src);  hipFree(d_dst);
  *dst_out = reinterpret_cast<char*>(h_dst);
  return len;
}
